#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
#include "/home/bpattabi/Work/bussAnal/filter/src/common.hpp"

#define TIMETHIS(command, startTimer, endTimer) \
  gettimeofday(startTimer, NULL);		\
  command;					\
  gettimeofday(endTimer, NULL);			\

#define NELEM 1e7
#define NTHREADS 512
#define NBLOCKS 512
#define CHUNK_SIZE 1e6


enum op_t {PLUS, MINUS};

typedef long long myType;

double getTimeDiff( 
		   struct timeval& start, 
		   struct timeval& end 
		    )
{
  return ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec)) / 1000 ;
}


// BP: Change input from integer to string
// BP: What happens if error in input? Handle this properly
// BP: Mus return status code of 0 if good; negative if bad 
template <typename T>
struct 
f1f2_functor
{
  char *op;
  f1f2_functor(char *in_op) : op(in_op) {}

  __host__ __device__
  T operator()(const T& x, const T& y) const
  {
    if ( strcmp(op, "+") == 0 ) { 
      return (x + y);
    }
    else if ( strcmp(op, "-") == 0 ) { 
      return (x - y);
    }
    else {
      // handle error case
    }
  }
};

template <typename T>
struct 
f1f2_functor
{
  int op;
  f1f2_functor(int in_op) : op(in_op) {}

  __host__ __device__
  T operator()(const T& x, const T& y) const
  {
    if(op == PLUS)
      return (x + y);
    else if(op == MINUS)
      return (x - y);
    else
      return (x + y);
  }
};

template <typename T, class BinaryFunction>
void 
transform_thrust(
		 T* f1, 
		 T* f2, 
		 T* f3, 
		 size_t nelem,
		 BinaryFunction bin_op
		 )
{
  //this device mem alloc takes finite time
  thrust::device_vector<T> d_f3 (nelem);

  thrust::device_vector<T> d_f1 (f1, f1 + nelem);// = h_f1;
  thrust::device_vector<T> d_f2 (f2, f2 + nelem);// = h_f2;

  thrust::transform(d_f1.begin(), d_f1.end(), d_f2.begin(), d_f3.begin(), bin_op);

  thrust::copy(d_f3.begin(), d_f3.end(), f3);
}


template <typename T, class BinaryFunction>
__global__ void 
gpu_add2_I4(
	    T* d_f1, 
	    T* d_f2, 
	    T* d_f3, 
	    size_t nelem
	    )
{
  //1-D thread index
  long long tid = blockDim.x * blockIdx.x + threadIdx.x;
  while(tid < nelem)
    {
      d_f3[tid] = d_f1[tid] + d_f2[tid];
      tid += blockDim.x * gridDim.x;
    }
}

template <typename T>
void 
transform_GPU_streams_wrap(
			   T* f1, 
			   T* f2, 
			   T* f3, 
			   size_t nelem, 
			   f1f2_functor<T> bin_op
			   )
{
  float elapsedTimeInMs = 0.0f;
  hipEvent_t cu_start, cu_stop;
  checkCudaErrors(hipEventCreate(&cu_start));
  checkCudaErrors(hipEventCreate(&cu_stop));

  checkCudaErrors(hipEventRecord(cu_start, 0));

  hipDeviceProp_t deviceProp;
  // This will pick the best possible CUDA capable device
  int devID = findCudaDevice(0, NULL);
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));



  // Kernel configuration, where a one-dimensional
  // grid and one-dimensional blocks are configured.
  dim3 dimGrid(NBLOCKS);
  dim3 dimBlock(NTHREADS);

  T *h_f1 = f1;
  T *h_f2 = f2;
  T *h_f3 = f3;
  /*
    T *h_f1;
    checkCudaErrors(hipHostMalloc( (void **) &h_f1, nelem * sizeof(T) ));
    T *h_f2;
    checkCudaErrors(hipHostMalloc( (void **) &h_f2, nelem * sizeof(T) ));
    T *h_f3;
    checkCudaErrors(hipHostMalloc( (void **) &h_f3, nelem * sizeof(T) ));
  */

  // allocate device memory for input array
  T *d_f1;
  checkCudaErrors(hipMalloc( (void **) &d_f1, nelem * sizeof(T) ));
  T *d_f2;
  checkCudaErrors(hipMalloc( (void **) &d_f2, nelem * sizeof(T) ));
  // allocate device memory for result
  T *d_f3;
  checkCudaErrors(hipMalloc( (void **) &d_f3,  nelem * sizeof(T) ));

  checkCudaErrors(hipEventRecord(cu_stop, 0));
  checkCudaErrors(hipDeviceSynchronize());                       \
  checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, cu_start, cu_stop));
  std::cout << "0 ->" << elapsedTimeInMs << std::endl;


  int chunks = nelem + (CHUNK_SIZE - 1) / CHUNK_SIZE;
  size_t last_chunk = nelem % (int)CHUNK_SIZE;

  //		for( int i = 0; i < chunks; i++ )
  {

    checkCudaErrors(hipEventRecord(cu_start, 0));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_f1, h_f1, nelem * sizeof(T), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_f2, h_f2, nelem * sizeof(T), hipMemcpyHostToDevice));

    checkCudaErrors(hipEventRecord(cu_stop, 0));
    checkCudaErrors(hipDeviceSynchronize());                       \
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, cu_start, cu_stop));
    std::cout << "1 ->" << elapsedTimeInMs << std::endl;


    checkCudaErrors(hipEventRecord(cu_start, 0));

    // execute the kernel
    transform_GPU<<< dimGrid, dimBlock >>>(d_f1, d_f2, d_f3, nelem, bin_op);
    hipDeviceSynchronize();

    checkCudaErrors(hipEventRecord(cu_stop, 0));
    checkCudaErrors(hipDeviceSynchronize());                       \
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, cu_start, cu_stop));
    std::cout << "2 ->" << elapsedTimeInMs << std::endl;



    // check if kernel execution generated and error
    //   getLastCudaError("Kernel execution failed");

    checkCudaErrors(hipEventRecord(cu_start, 0));

    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_f3, d_f3, nelem * sizeof(T), hipMemcpyDeviceToHost));

    checkCudaErrors(hipEventRecord(cu_stop, 0));
    checkCudaErrors(hipDeviceSynchronize());                       \
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, cu_start, cu_stop));
    std::cout << "3 ->" << elapsedTimeInMs << std::endl;
  }

  // free device memory
  hipFree(d_f1);
  hipFree(d_f2);
  hipFree(d_f3);
}


template <typename T>
void 
transform_GPU_wrap(
		   T* h_f1, 
		   T* h_f2, 
		   T* h_f3, 
		   size_t nelem, 
		   char *op
		   )
{
  hipDeviceProp_t deviceProp;
  // This will pick the best possible CUDA capable device
  int devID = findCudaDevice(0, NULL);
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

  // Kernel configuration, where a one-dimensional
  // grid and one-dimensional blocks are configured.
  dim3 dimGrid(NBLOCKS);
  dim3 dimBlock(NTHREADS);

  // allocate device memory for input array
  T *d_f1;
  checkCudaErrors(hipMalloc( (void **) &d_f1, nelem * sizeof(T) ));
  T *d_f2;
  checkCudaErrors(hipMalloc( (void **) &d_f2, nelem * sizeof(T) ));
  // allocate device memory for result
  T *d_f3;
  checkCudaErrors(hipMalloc( (void **) &d_f3,  nelem * sizeof(T) ));

  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_f1, h_f1, nelem * sizeof(T), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_f2, h_f2, nelem * sizeof(T), hipMemcpyHostToDevice));

  // execute the kernel
  if ( strcmp(op, "+") == 0 ) { 
    gpu_add2_I4<<< dimGrid, dimBlock >>>(d_f1, d_f2, d_f3, nelem);
  }
  else {
  }

  // check if kernel execution generated and error
  getLastCudaError("Kernel execution failed");

  // copy result from device to host
  checkCudaErrors(hipMemcpy(h_f3, d_f3, nelem * sizeof(T), hipMemcpyDeviceToHost));

  // free device memory
  hipFree(d_f1);
  hipFree(d_f2);
  hipFree(d_f3);
}

int main(void)
{

  struct timeval start, end;

  // START: Allocate and initialize code 
  myType* f1 = (myType*) malloc(NELEM * sizeof(myType));
  myType* f2 = (myType*) malloc(NELEM * sizeof(myType));
  myType* f3 = (myType*) malloc(NELEM * sizeof(myType));

  for ( long long i=0; i<NELEM; i++) {
    f1[i] = i;
    f2[i] = NELEM - i - 1;
  }
  // STOP: Allocate and initialize code 

  // VARIANT 1: THRUST

  gettimeofday(&start, NULL);
  //	transform_thrust(f1, f2, f3, NELEM, f1f2_functor<myType>(0));
  transform_GPU_wrap(f1, f2, f3, NELEM, f1f2_functor<myType>(0));
  transform_GPU_streams_wrap(f1, f2, f3, NELEM, f1f2_functor<myType>(0));
  gettimeofday(&end, NULL);
  std::cout << "gpu add = " << getTimeDiff(start, end) << "ms" << std::endl;


  //_CILK_FOR equivalent

  /*
    TIMETHIS( for(long long i=0; i<(long long)(NELEM); i++, y[i] = 2.0 * x[i] + y[i]), &start, &end );

    cout << "cpu add = " << getTimeDiff(start, end) << endl;
  */

  //SEQUENTIAL

  thrust::host_vector<myType> h_f1 (f1, f1 + (size_t)NELEM);
  thrust::host_vector<myType> h_f2 (f2, f2 + (size_t)NELEM);
  thrust::host_vector<myType> h_f3_cpu(NELEM);
  //	TIMETHIS( std::transform (h_f1.begin(), h_f1.end(), h_f2.begin(), h_f3_cpu.begin(), f1f2_functor<myType>()), &start, &end );

  //	std::cout << "cpu add = " << getTimeDiff(start, end) << "ms" << std::endl;


  TIMETHIS( for(long long i=0; i<NELEM; h_f3_cpu[i] = f1[i] + f2[i], i++), &start, &end );

  std::cout << "cpu add = " << getTimeDiff(start, end) << "ms" << std::endl;

  for(long long i=0; i < NELEM; i++)
    {
      if(f3[i] != h_f3_cpu[i])
	{
	  std::cout << "Results dont match!!! at " << i << " cpu: " << h_f3_cpu[i] << " gpu: " << f3[i] << std::endl;
	  break;
	}
    }

  free(f1);
  free(f2);
  free(f3);

  return 0;
}
